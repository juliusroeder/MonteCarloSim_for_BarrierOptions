#include "hip/hip_runtime.h"
//
// Created by Julius Roeder on 04/10/2022.
//

#include "Simulator.cuh"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include <random>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <iostream>

#define BILLION 1000000000L
#define MILLION 1000000L

__global__ void  down_and_out_call_kernel(
        const float* rand_num,
        const float K,
        const float B,
        const float S0,
        const float sigma,
        const float mu,
        const float dt,
        const unsigned N_PATH,
        const unsigned N_STEP,
        float * d_answer){

    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int index_rand_num = index * N_STEP ;

    __shared__ float partial_payoff;
    if (threadIdx.x == 0) partial_payoff=0;
    if (index == 0) *d_answer = 0;
    __syncthreads();

    // simulation part
    if(index < N_PATH) {
        float current_price = S0;
        unsigned int n = 0;
        while (n < N_STEP && current_price > B) {
            current_price += mu * current_price * dt + sigma * current_price * rand_num[index_rand_num];
            index_rand_num++;
            n++;
        }
        (current_price > K ? atomicAdd(&partial_payoff,current_price - K) : 0);
    }

    __syncthreads(); //make sure all threads are done
    if (threadIdx.x==0) atomicAdd(d_answer, partial_payoff); //thread 0 in a block adds the block payoff to the global payoff
}

Simulator::Simulator(simulation_params *simParams) {
    params = simParams;



}

Simulator::~Simulator() {
    hipFree(rand_num);
    hipFree(d_answer);
}

void Simulator::prepGpu(){
    ////  allocate space on GPU
    hipMalloc(&d_answer, sizeof(float));
    hipMalloc((void**)&rand_num, params->m_nRandNum * sizeof(float));

    ////  generate random numbers
    hiprandGenerator_t GPU_generator;
    hiprandCreateGenerator(&GPU_generator, HIPRAND_RNG_PSEUDO_MTGP32);
    hiprandSetPseudoRandomGeneratorSeed(GPU_generator, 3567357ULL);
    hiprandGenerateNormal(GPU_generator, rand_num, params->m_nRandNum, 0.0f, params->m_sqrt_dt);
}

void Simulator::prepCpu(){
    ////  CPU Version
    std::default_random_engine CPU_generator;
    std::normal_distribution<float> distribution (0, params->m_sqrt_dt);

    auto gen = [&distribution, &CPU_generator](){
        return distribution(CPU_generator);
    };

    cpu_rands.resize(params->m_nRandNum);
    std::generate(begin(cpu_rands), end(cpu_rands), gen);
}

void Simulator::downAndOutCallKernel(){
    int numBlocks = ceil((float)params->c_nPath/512.0f);
    down_and_out_call_kernel<<<numBlocks, 512>>>(rand_num, params->c_K, params->c_B, params->c_S0, params->c_sigma,
                                                 params->c_mu, params->m_dt, params->c_nPath, params->c_nSteps, d_answer);
}

void Simulator::runGpuSim(){
    ////  call Kernel
    if (params->c_type == "down_and_out_call") {
        downAndOutCallKernel();
    }else if (params->c_type.empty()){
        std::cout << "No option type specified." << std::endl;
        throw;
    }

    hipMemcpy(&h_answer, d_answer, sizeof(float), hipMemcpyDeviceToHost);
    std::cout << "GPU result " << params->m_exponent * h_answer/params->c_nPath << std::endl;
}

void Simulator::runCpuSim(){ //down_and_out call option

    float payoff = 0;
    unsigned ii = 0;
    for (unsigned i=0; i < params->c_nPath; i++){
        int j = 0;
        float curr_value = params->c_S0;
        ii = i * params->c_nSteps; // need to actually start at the right random value
        while (j < params->c_nSteps && curr_value > params->c_B){
            curr_value = curr_value + params->c_mu * curr_value * params->m_dt + params->c_sigma * curr_value * cpu_rands[ii];
            j++;
            ii++;
        }
        payoff += params->m_exponent * (curr_value > params->c_K ? curr_value - params->c_K : 0);
    }

    std::cout << "CPU result " << payoff/(float)params->c_nPath << std::endl;
}

void Simulator::runSimulation(){
    ////  Profiling
    unsigned long diff_cpu = 0, diff_gpu = 0;
    struct timespec start{}, end{};
    clock_gettime(CLOCK_MONOTONIC, &start);

//  Start GPU Monte Carlo Sim
    prepGpu();
    runGpuSim();

    ////  Profiling
    clock_gettime(CLOCK_MONOTONIC, &end);
    diff_gpu = BILLION * (end.tv_sec - start.tv_sec) + end.tv_nsec - start.tv_nsec;
    std::cout << "GPU Time Taken in ms " << (double)diff_gpu/MILLION << std::endl;

    ////  Profiling
    clock_gettime(CLOCK_MONOTONIC, &start);

// Start CPU Monte Carlo Sim
    prepCpu();
    runCpuSim();

    ////  Profiling
    clock_gettime(CLOCK_MONOTONIC, &end);
    diff_cpu = BILLION * (end.tv_sec - start.tv_sec) + end.tv_nsec - start.tv_nsec;
    std::cout << "CPU Time Taken in ms " << (double)diff_cpu/MILLION << std::endl;

    std::cout << "Speedup: " << (double)diff_cpu/(double)diff_gpu << std::endl;
}







